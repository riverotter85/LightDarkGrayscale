#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <filesystem>
#include <list>
#include <exception>
#include <vector>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>

#include "CudaImage.hpp"

using namespace std;

__global__ void applyLightDarkGrayscale(uchar *d_r, uchar *d_g, uchar *d_b, uchar *d_bright, uchar *d_dark, uchar *d_grayscale, int brightPercentage, int darkPercentage) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                    + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                            + (threadIdx.z * (blockDim.x * blockDim.y))
                            + (threadIdx.y * blockDim.x) + threadIdx.x;
    
    // Bright
    float modifier = 1 + brightPercentage * 0.01;
    d_bright[threadId] = (d_r[threadId] + d_g[threadId] + d_b[threadId]) * modifier;

    // Dark
    modifier = 1 - darkPercentage * 0.01;
    d_dark[threadId] = (d_r[threadId] + d_g[threadId] + d_b[threadId]) * modifier;

    // Grayscale
    d_grayscale[threadId] = (d_r[threadId] + d_g[threadId] + d_b[threadId]) / 3;

    __syncthreads();
}

__host__ void executeKernel(CudaImage *ci, int brightPercentage, int darkPercentage, int threadsPerBlock) {
    cout << "Running kernel for LightDarkGrayscale...\n";

    const int blockZSize = 4;
    const int gridCols = min(ci->cols / (threadsPerBlock * 4), 1);
    dim3 grid(ci->rows, gridCols, 1);
    dim3 block(1, threadsPerBlock, blockZSize);

    // Kernel code
    applyLightDarkGrayscale<<<grid, block>>>(ci->d_r, ci->d_g, ci->d_b, ci->d_bright, ci->d_dark, ci->d_grayscale, brightPercentage, darkPercentage);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch bright kernel: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    cout << "Done.\n";
}

__host__ int promptInputPercentage(string prompt) {
    int response;
    do {
        cout << prompt << ": ";
        cin >> response;
    } while (response < 0 || response > 100);

    return response;
}

__host__ int main(int argc, char **argv) {
    const int threadsPerBlock = 8;
    vector<string> filepaths;
    vector<CudaImage *> cudaImages;

    // Get a list of filenames
    string filepath = "./data";
    for (const auto &entry : filesystem::directory_iterator(filepath)) {
        filepaths.push_back(entry.path());
    }

    try {
        // Prompt user for percentage to brighten images by
        int brightPercentage = promptInputPercentage("Enter percentage to brighten first image");

        // Prompt user for percentage to darken images by
        int darkPercentage = promptInputPercentage("Enter percentage to darken second image");

        // Iterate through each file
        CudaImage *ci = NULL;
        for (string path : filepaths) {
            ci = createCudaImage(path);
            copyFromHostToDevice(ci);

            // Execute kernel
            executeKernel(ci, brightPercentage, darkPercentage, threadsPerBlock);

            cudaImages.push_back(ci);
        }

        // Synchronize all threads
        // __syncthreads();

        // Now that our data operations are finished, commence with mapping to output files
        for (int i = 0; i < filepaths.size(); ++i) {
            copyFromDeviceToHost(cudaImages[i]);

            mapBrightImage(cudaImages[i], "bright_" + filepaths[i]);
            mapDarkImage(cudaImages[i], "dark_" + filepaths[i]);
            mapGrayscaleImage(cudaImages[i], "grayscale_" + filepaths[i]);

            destroyCudaImage(cudaImages[i]);
        }

    } catch (Exception &e) {
        fprintf(stderr, "Caught exception: %s\n", e.what());
        exit(EXIT_FAILURE);
    }

    exit(EXIT_SUCCESS);
}